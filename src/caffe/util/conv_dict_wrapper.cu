#include "hip/hip_runtime.h"
#include "caffe/util/conv_dict_wrapper.hpp"
#include "caffe/util/math_functions.hpp"
#include "thrust/execution_policy.h"
#include "thrust/count.h"
#include "thrust/copy.h"
#include "thrust/find.h"
#include "thrust/device_vector.h"
#include "thrust/device_ptr.h"
#include "thrust/functional.h"
#include "thrust/equal.h"

namespace caffe {
template <typename Dtype>
__global__ void set_shifted_kernel(const int vec_len, int n, int m, const Dtype *x,
    Dtype *y) {
    CUDA_KERNEL_LOOP(index, vec_len) {
        int conv_row = index / (n*m);
        int conv_col = index % (n*m);
        int block = conv_col / m;
        int col = conv_col % m;
        conv_row = conv_row > (n-1) ? (n-1) : conv_row;
        int row = (conv_row - block + n) % n;
        y[index] = x[row * m + col];
    }
}

template <typename Dtype>
__global__ void index_shifted_kernel(const int vec_len, int n, int m, int N, int *y) {
    CUDA_KERNEL_LOOP(index, vec_len) {
        int i = index / (n*m);
        int j_index = index % (n*m);
        int col = j_index % m;
        int j = j_index / m;
        int block = i >= n ? (i + j - n + 1) : (i >= j ? j : (N-n+j));
        y[index] = block * m + col;
    }
}


template <typename Dtype>
__global__ void index_inc_kernel(const int vec_len, const int inc, int *y) {
    CUDA_KERNEL_LOOP(index, vec_len) {
        y[index] = index * inc;
    }
}

// Standalone funtion to create a zero-based sparse CSR format convolutional dictionary
// from a dense local dictionary Dl.
// Dl has row n and column m, and d_values and d_columns have length n*m*N, d_ptrB have
// length N + 1.
template <typename Dtype>
void make_conv_dict_gpu(const int n, const int m, const Dtype *d_Dl, const int N,
    CSCParameter::Boundary boundary, Dtype *d_values, int *d_columns, int *d_ptrB) {
    CHECK_EQ(boundary, CSCParameter::CIRCULANT_BACK) 
        << "Only circulant back boundary is supported!";
    set_shifted_kernel<Dtype><<<CAFFE_GET_BLOCKS(N*m*n), CAFFE_CUDA_NUM_THREADS>>>(
        N*m*n, n, m, d_Dl, d_values);
    index_shifted_kernel<Dtype><<<CAFFE_GET_BLOCKS(N*m*n), CAFFE_CUDA_NUM_THREADS>>>(
        N*m*n, n, m, N, d_columns);
    index_inc_kernel<Dtype><<<CAFFE_GET_BLOCKS(N+1), CAFFE_CUDA_NUM_THREADS>>>(
        N+1, n*m, d_ptrB);
}

// The creation of identity is in cu file because it requires a iota like kernel.
template <typename Dtype>
CSRWrapper<Dtype> &CSRWrapper<Dtype>::identity() {
    CHECK_EQ(row(), col());
    CHECK_EQ(row(), nnz());
    caffe_gpu_set(nnz(), Dtype(1), mutable_values());
    index_inc_kernel<Dtype><<<CAFFE_GET_BLOCKS(nnz()), CAFFE_CUDA_NUM_THREADS>>>(
        nnz(), 1, mutable_columns());
    index_inc_kernel<Dtype><<<CAFFE_GET_BLOCKS(row()+1), CAFFE_CUDA_NUM_THREADS>>>(
        row()+1, 1, mutable_ptrB());
    return *this;
}

struct InverseIndex {
    explicit InverseIndex(int nnz, const int *d_inds, int not_found)
        : nnz_(nnz), not_found_(not_found),
        inds_(thrust::device_pointer_cast(d_inds)) {}
    __host__ __device__ int operator()(const int &x) const {
        thrust::device_ptr<const int> found = thrust::find(thrust::device, inds_, inds_+nnz_, x);
        return (found != inds_+nnz_ ? found-inds_ : not_found_);
    }
    int nnz_;
    int not_found_;
    thrust::device_ptr<const int> inds_;
};

struct NonNegative {
    __host__ __device__ bool operator()(int x) {
        return x >= 0;
    }
};

template <typename Dtype>
shared_ptr<CSRWrapper<Dtype> > CSRWrapper<Dtype>::clip_columns_gpu_(int nnz, const int *d_inds) {
    CHECK_GE(nnz_, nnz);
    thrust::device_ptr<const int> thrust_inds = thrust::device_pointer_cast(d_inds);
    CHECK(thrust::equal(thrust::device, thrust_inds, thrust_inds+nnz-1, thrust_inds+1, thrust::less<int>()));

    shared_ptr<CSRWrapper<Dtype> > clipped(new CSRWrapper<Dtype>(handle_, r_, nnz, -1));
    thrust::device_vector<int> stencil(nnz_, -1);
    thrust::device_ptr<Dtype> thrust_values = thrust::device_pointer_cast(mutable_values());
    thrust::device_ptr<int> thrust_columns = thrust::device_pointer_cast(mutable_columns());
    InverseIndex inverse_index(nnz, d_inds, -1);
    NonNegative non_negative;
    thrust::transform(thrust::device, thrust_columns, thrust_columns + nnz_, stencil.begin(), inverse_index);
    clipped->mutable_cpu_ptrB()[0] = 0;
    for (int i = 0; i < r_; ++i) {
        clipped->mutable_cpu_ptrB()[i+1] = clipped->mutable_cpu_ptrB()[i] +
            thrust::count_if(thrust::device, stencil.begin() + cpu_ptrB()[i],
                stencil.begin() + cpu_ptrE()[i], non_negative);
    }
    clipped->set_nnz(clipped->mutable_cpu_ptrB()[r_]);
    thrust::copy_if(thrust::device, thrust_values, thrust_values + nnz_, stencil.begin(),
        thrust::device_pointer_cast(clipped->mutable_values()), non_negative);
    thrust::copy_if(thrust::device, stencil.begin(), stencil.end(),
        thrust::device_pointer_cast(clipped->mutable_columns()), non_negative);
    return clipped;
}

// require instaniation
template CSRWrapper<float>  &CSRWrapper<float>::identity();
template CSRWrapper<double> &CSRWrapper<double>::identity();
template shared_ptr<CSRWrapper<float> > CSRWrapper<float>::clip_columns_gpu_(int nnz, const int *d_inds);
template shared_ptr<CSRWrapper<double> > CSRWrapper<double>::clip_columns_gpu_(int nnz, const int *d_inds);

template void make_conv_dict_gpu<float>(const int n, const int m, const float *d_Dl, const int N,
    CSCParameter::Boundary boundary, float *d_values, int *d_columns, int *d_ptrB);
template void make_conv_dict_gpu<double>(const int n, const int m, const double *d_Dl, const int N,
    CSCParameter::Boundary boundary, double *d_values, int *d_columns, int *d_ptrB);


} // namespace caffe
